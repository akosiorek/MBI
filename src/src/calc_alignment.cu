#include "hip/hip_runtime.h"
/*
 * sample.cu
 *
 *  Created on: Jan 17, 2015
 *      Author: Adam Kosiorek
 */

#include "calc_alignment.h"

__global__ void calc_alignment_GPU_impl(int* x, int* y, int* z, int nX, int nY,
		int nZ, int* cube) {

}

Cube calc_alignment_GPU(const Sequence& seq) {

	std::vector<int*> data(seq.size());
	int totalSize = 1;
	for (int i = 0; i < data.size(); ++i) {
		hipMalloc((void**) data[i], seq[i].size());
		hipMemcpy(data[i], &seq[i][0], seq[i].size() * sizeof(int),
				hipMemcpyHostToDevice);
		totalSize *= seq[i].size();
	}
	int* cube = NULL;
	hipMalloc((void**) cube, totalSize);

	calc_alignment_GPU_impl<<<1, 1>>>(data[0], data[1], data[2], seq[0].size(), seq[1].size(), seq[2].size(), cube);

	Cube output(seq[0].size(),
			std::vector<std::vector<int> >(seq[1].size(),
					std::vector<int>(seq[2].size())));

	for (int x = 0; x < seq[0].size(); ++x) {
		for (int y = 0; y < seq[1].size(); ++y) {
			hipMemcpy(&output[x][y][0], cube, seq[2].size() * sizeof(int),
					hipMemcpyDeviceToHost);
			cube += seq[2].size();
		}
	}

	return output;
}
